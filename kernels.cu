#include "hip/hip_runtime.h"
#include "kernels.h"


using namespace kernels;

#define CUDA_BLOCK_SIZE 256

namespace {

__global__ void get_centers_kernel(float* points, float* centers,
                                   int batch_size, int sample_size) {
    int batch = blockIdx.x * blockDim.x + threadIdx.x;
    while (batch < batch_size) {
        float x = 0, y = 0;
        for (int i = 0; i < sample_size; ++i) {
            x += points[batch * sample_size * 2 + i * 2];
            y += points[batch * sample_size * 2 + i * 2 + 1];
        }
        centers[batch * 2] = x / sample_size;
        centers[batch * 2 + 1] = y / sample_size;
        batch += gridDim.x * blockDim.x;
    }
}

__global__ void fill_param_kernel(float* points, float* centers, double* A,
                                  double* b, int batch_size, int sample_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < batch_size * sample_size) {
        int batch_id = index / sample_size;
        int sample_id = index - batch_id * sample_size;
        float x = points[index * 2] - centers[batch_id * 2];
        float y = points[index * 2 + 1] - centers[batch_id * 2 + 1];
        int sample_offset = batch_id * sample_size * 5 + sample_id;
        A[sample_offset] = -(double)x * (double)x;
        A[sample_offset + sample_size] = -(double)y * (double)y;
        A[sample_offset + sample_size * 2] = -(double)x * (double)y;
        A[sample_offset + sample_size * 3] = x;
        A[sample_offset + sample_size * 4] = y;
        b[index] = 10000.0;

        index += gridDim.x * blockDim.x;
    }

}

}

void kernels::get_centers(float* points, float* centers, int batch_size,
                          int sample_size) {
    int grid = (batch_size + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
    get_centers_kernel<<<grid, CUDA_BLOCK_SIZE>>>(points, centers, batch_size,
                                                  sample_size);
}

void kernels::fill_param(float* points, float* centers, double* A, double* b,
                         double x, int batch_size, int sample_size) {
    return;
}

}

